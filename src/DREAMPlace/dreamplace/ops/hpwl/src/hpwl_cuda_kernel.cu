#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/Msg.h"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
void printArray(const T* x, const int n, const char* str)
{
    printf("%s[%d] = ", str, n);
    T* host_x = (T*)malloc(n*sizeof(T));
    if (host_x == NULL)
    {
        printf("failed to allocate memory on CPU\n");
        return;
    }
    hipMemcpy(host_x, x, n*sizeof(T), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i)
    {
        printf("%g ", double(host_x[i]));
    }
    printf("\n");

    free(host_x);
}

template <typename T>
void printScalar(const T& x, const char* str)
{
    printf("%s = ", str);
    T* host_x = (T*)malloc(sizeof(T));
    if (host_x == NULL)
    {
        printf("failed to allocate memory on CPU\n");
        return;
    }
    hipMemcpy(host_x, &x, sizeof(T), hipMemcpyDeviceToHost);
    printf("%g\n", double(*host_x));

    free(host_x);
}

template <typename T>
__global__ void fillArray(T* x, const int n, const T v)
{
    //for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    int i  = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        x[i] = v;
    }
}

template <typename T>
__global__ void computeHPWL(
        const T* x,
        const int* flat_netpin,
        const int* netpin_start,
        const unsigned char* net_mask,
        int num_nets,
        T* partial_hpwl
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets)
    {
        T max_x = -FLT_MAX;
        T min_x = FLT_MAX;

        if (net_mask[i])
        {
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                min_x = min(min_x, x[flat_netpin[j]]);
                max_x = max(max_x, x[flat_netpin[j]]);
            }
            partial_hpwl[i] = max_x-min_x;
        }
        else
        {
            partial_hpwl[i] = 0;
        }
    }
}

template <typename T>
int computeHPWLCudaLauncher(
        const T* x, const T* y,
        const int* flat_netpin,
        const int* netpin_start,
        const unsigned char* net_mask,
        int num_nets,
        T* partial_hpwl
        )
{
    const int thread_count = 512;
    const int block_count_nets = (num_nets + thread_count - 1) / thread_count;

    hipError_t status;
    hipStream_t stream_y;
    status = hipStreamCreate(&stream_y);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_y\n");
        fflush(stdout);
        return 1;
    }

    computeHPWL<<<block_count_nets, thread_count>>>(
            x,
            flat_netpin,
            netpin_start,
            net_mask,
            num_nets,
            partial_hpwl
            );

    computeHPWL<<<block_count_nets, thread_count, 0, stream_y>>>(
            y,
            flat_netpin,
            netpin_start,
            net_mask,
            num_nets,
            partial_hpwl+num_nets
            );

    /* destroy stream */
    status = hipStreamDestroy(stream_y);
    if (status != hipSuccess)
    {
        printf("stream_y destroy failed\n");
        fflush(stdout);
        return 1;
    }

    //printArray(partial_hpwl, num_nets, "partial_hpwl");

    // I move out the summation to use ATen
    // significant speedup is observed
    //sumArray<<<1, 1>>>(partial_hpwl, num_nets, hpwl);

    return 0;
}

// manually instantiate the template function
#define REGISTER_KERNEL_LAUNCHER(type) \
    int instantiateComputeHPWLLauncher(\
        const type* x, const type* y, \
        const int* flat_netpin, \
        const int* netpin_start, \
        const unsigned char* net_mask, \
        int num_nets, \
        type* partial_hpwl \
        ) \
    { \
        return computeHPWLCudaLauncher(x, y, \
                flat_netpin, \
                netpin_start, \
                net_mask, \
                num_nets, \
                partial_hpwl \
                ); \
    }

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
